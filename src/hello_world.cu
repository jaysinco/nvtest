#include "hip/hip_runtime.h"
#include "fwd.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void test_print() { printf("Hello World from GPU!\n"); }

int hello_world(int argc, char** argv)
{
    test_print<<<1, 1>>>();
    hipDeviceReset();
    return 0;
}